#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <renderer.hpp>
#include <time.h>



__global__ void _construct(Scene* scene) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        scene->volumes[0] = new Sphere(glm::vec3(-2, 0, -4), 0.5);
        scene->volumes[1] = new Sphere(glm::vec3(0, 0, -8), 0.75);
        scene->volumes[2] = new Sphere(glm::vec3(2, 0, -4), 1);
        scene->volumes[3] = new Plane(-1);
    }
}

__global__ void _render_init(uint width, uint height, hiprandState *randState) {
    const uint i = threadIdx.x + blockIdx.x * blockDim.x;
    const uint j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= width || j >= height) return;

    const uint pixel = j * width + i;

    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1234, pixel, 0, &randState[pixel]);
}

__global__ void _render(float* fb, uint width, uint height, 
    const Scene* scene, const Camera* cam, hiprandState *randState) {
    const uint i = threadIdx.x + blockIdx.x * blockDim.x;
    const uint j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= width || j >= height) return;

    const uint pixel = (j * width + i) * 3;
    hiprandState localRandState = randState[pixel];
    glm::vec3 color(0, 0, 0);
    for (uint s = 0; s < cam->samplesPerPixel; ++s) {
        const glm::vec2 uv(
            (float(i) + hiprand_uniform(&localRandState)) / float(width), 
            (float(j) + hiprand_uniform(&localRandState)) / float(height));
        const Ray ray = cam->ray(uv);
        color += scene->colorAt(ray, &localRandState);
    }
    color /= float(cam->samplesPerPixel);
    fb[pixel + 0] = sqrt(color.x);
    fb[pixel + 1] = sqrt(color.y);
    fb[pixel + 2] = sqrt(color.z);
}

Renderer::Renderer(const uint width, const uint height) :
    width(width), height(height), framebufferLen(width * height * sizeof(float) * 3),
    camera(glm::vec3(0, 0, 0), float(width) / float(height), 1.0f) {
    catchErr(hipMallocManaged((void **)&framebuffer, framebufferLen));
}

Renderer::~Renderer() {
    catchErr(hipFree(framebuffer));
}

void Renderer::render(float* dest) {
    printf("Rendering %ux%u image...\n", width, height);
    const uint blockSize = 16;

    //Timing clock
    clock_t start, stop;
    start = clock();

    //CUDA Random state
    hiprandState* randState;
    catchErr(hipMalloc((void**)&randState, width * height * sizeof(hiprandState)));
    
    //Construct scene
    Scene scene(4);
    Scene* _scene;
    catchErr(hipMalloc((void**)&_scene, sizeof(Scene)));
    catchErr(hipMemcpy(_scene, &scene, sizeof(Scene), hipMemcpyHostToDevice));
    puts("Constructing scene..");
    _construct<<<1, 1>>>(_scene);
    catchErr(hipGetLastError());
    catchErr(hipDeviceSynchronize());

    //Copy memory
    Camera* _camera;
    catchErr(hipMalloc((void**)&_camera, sizeof(Camera)));
    catchErr(hipMemcpy(_camera, &this->camera, sizeof(Camera), hipMemcpyHostToDevice));

    //Render to framebuffer
    dim3 blocks(width / blockSize + 1, height / blockSize + 1);
    dim3 threads(blockSize, blockSize);

    puts("Initializing render...");
    _render_init<<<blocks, threads>>>(width, height, randState);
    catchErr(hipGetLastError());
    catchErr(hipDeviceSynchronize());

    puts("Rendering...");
    _render<<<blocks, threads>>>(framebuffer, width, height, _scene, _camera, randState);
    catchErr(hipGetLastError());
    catchErr(hipDeviceSynchronize());

    stop = clock();
    double seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("Finished in %lf seconds\n", seconds);

    //Copy out
    catchErr(hipMemcpy(
        (void *)dest, (void *)framebuffer, 
        framebufferLen, hipMemcpyDeviceToHost));
    
    //Free memory
    catchErr(hipFree(_camera));
    catchErr(hipFree(_scene));
}

