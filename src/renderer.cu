#include "hip/hip_runtime.h"
#include <iostream>
#include <renderer.hpp>
#include <time.h>



__global__ void _construct(Volume** volumes) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        volumes[1] = new Sphere(glm::vec3(-2, 0, -4), 0.5);
        volumes[0] = new Sphere(glm::vec3(0, 0, -4), 0.75);
        volumes[2] = new Sphere(glm::vec3(2, 0, -4), 1);
    }
}

__global__ void _render(float* fb, uint width, uint height, 
    const Scene* scene, const Camera* cam, Volume** volumes, size_t nvolumes) {
    const uint i = threadIdx.x + blockIdx.x * blockDim.x;
    const uint j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= width || j >= height) return;

    const uint pixel = (j * width + i) * 3;
    const glm::vec2 uv(float(i) / float(width), float(j) / float(height));
    const Ray ray = cam->ray(uv);
    const glm::vec3 color = scene->colorAt(ray, volumes, nvolumes);
    fb[pixel + 0] = color.x;
    fb[pixel + 1] = color.y;
    fb[pixel + 2] = color.z;
}

Renderer::Renderer(const uint width, const uint height) :
    width(width), height(height), framebufferLen(width * height * sizeof(float) * 3),
    camera(glm::vec3(0, 0, 0), float(width) / float(height), 1.0f) {
    catchErr(hipMallocManaged((void **)&framebuffer, framebufferLen));
}

Renderer::~Renderer() {
    catchErr(hipFree(framebuffer));
}

void Renderer::render(float* dest) {
    printf("Rendering %ux%u image...\n", width, height);
    const uint blockSize = 16;

    //Timing clock
    clock_t start, stop;
    start = clock();

    //Copy memory
    Camera* _camera;
    catchErr(hipMalloc((void**)&_camera, sizeof(Camera)));
    catchErr(hipMemcpy(_camera, &this->camera, sizeof(Camera), hipMemcpyHostToDevice));
    Scene* _scene;
    catchErr(hipMalloc((void**)&_scene, sizeof(Scene)));
    catchErr(hipMemcpy(_scene, &this->scene, sizeof(Scene), hipMemcpyHostToDevice));
    
    //Construct scene
    size_t nvolumes = 3;
    Volume** _volumes;
    catchErr(hipMalloc((void**)&_volumes, nvolumes * sizeof(Volume *)));
    _construct<<<1, 1>>>(_volumes);
    catchErr(hipGetLastError());
    catchErr(hipDeviceSynchronize());

    //Render to framebuffer
    dim3 blocks(width / blockSize + 1, height / blockSize + 1);
    dim3 threads(blockSize, blockSize);
    _render<<<blocks, threads>>>(framebuffer, width, height, _scene, _camera, _volumes, nvolumes);

    //Catch errors, print time
    catchErr(hipGetLastError());
    catchErr(hipDeviceSynchronize());
    stop = clock();
    double seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("Finished in %lf seconds\n", seconds);

    //Copy out
    catchErr(hipMemcpy(
        (void *)dest, (void *)framebuffer, 
        framebufferLen, hipMemcpyDeviceToHost));
    
    //Free memory
    catchErr(hipFree(_camera));
    catchErr(hipFree(_scene));
}

