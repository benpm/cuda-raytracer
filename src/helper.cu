#include <helper.hpp>

void checkCUDA(hipError_t result, char const *const func, 
    const char *const file, int const line) {
    if (result) {
        std::cerr << "! CUDA ERROR: " << hipGetErrorName(result) << std::endl;
        std::cerr << "\t" << hipGetErrorString(result) << std::endl;
        std::cerr << "\tat " << file << ":" << line << " " << func << std::endl;
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}