#include "hip/hip_runtime.h"
#include <volume.hpp>

#define T_MIN 0.001f

__device__ Sphere::Sphere(Material* mat, const glm::vec3& pos, float radius)
    : radius(radius), pos(pos), mat(mat) {
}

__device__ bool Sphere::intersect(const Ray& ray, float minT, float maxT, Hit& hit) const {
    const glm::vec3 oc = ray.a - pos;
    const float a = glm::dot(ray.b, ray.b);
    const float b = glm::dot(oc, ray.b);
    const float c = glm::dot(oc, oc) - radius * radius;
    const float discriminant = (b * b) - (a * c);
    if (discriminant > 0) {
        float t = (-b - sqrt(discriminant)) / a;
        if (t > minT && t < maxT) {
            hit.t = t;
            hit.point = ray.pointAtTime(t);
            hit.normal = glm::normalize((hit.point - pos) / radius);
            return true;
        }
        t = (-b + sqrt(discriminant)) / a;
        if (t > minT && t < maxT) {
            hit.t = t;
            hit.point = ray.pointAtTime(t);
            hit.normal = glm::normalize((hit.point - pos) / radius);
            return true;
        }
    }
    return false;
}

__device__ Material* Sphere::getMat() const {
    return this->mat;
}

__device__ Plane::Plane(Material* mat, float height)
    : height(height), mat(mat) {
}

__device__ bool Plane::intersect(const Ray& ray, float minT, float maxT, Hit& hit) const {
    float t = (height - ray.a.y) / ray.b.y;
    if (t > minT && t < maxT) {
        hit.t = t;
        hit.point = ray.pointAtTime(hit.t);
        hit.normal = glm::vec3(0, 1, 0);
        return true;
    }

    return false;
}

__device__ Material* Plane::getMat() const {
    return this->mat;
}