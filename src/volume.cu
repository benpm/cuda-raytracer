#include "hip/hip_runtime.h"
#include <volume.hpp>

__device__ Sphere::Sphere(const glm::vec3& pos, float radius) : radius(radius), pos(pos) {
}

__device__ bool Sphere::intersect(const Ray& ray, Hit& hit) const {
    const glm::vec3 oc = ray.a - pos;
    const float a = glm::dot(ray.b, ray.b);
    const float b = glm::dot(oc, ray.b);
    const float c = glm::dot(oc, oc) - radius * radius;
    const float discriminant = (b * b) - (a * c);
    if (discriminant > 0) {
        float t = (-b - sqrt(discriminant)) / a;
        if (t > 0) {
            hit.t = t;
            hit.point = ray.pointAtTime(t);
            hit.normal = glm::normalize((hit.point - pos) / radius);
            return true;
        }
        t = (-b + sqrt(discriminant)) / a;
        if (t > 0) {
            hit.t = t;
            hit.point = ray.pointAtTime(t);
            hit.normal = glm::normalize((hit.point - pos) / radius);
            return true;
        }
    }
    return false;
}

__device__ Plane::Plane(float height) : height(height) {
}

__device__ bool Plane::intersect(const Ray& ray, Hit& hit) const {
    hit.t = (height - ray.a.y) / ray.b.y;
    if (hit.t > 0) {
        hit.t = -ray.a.y / ray.b.y;
        hit.point = ray.pointAtTime(hit.t);
        hit.normal = glm::vec3(0, 1, 0);
        return true;
    }

    return false;
}