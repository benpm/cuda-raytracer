#include <scene.hpp>

#define RANDVEC3(R) glm::vec3(hiprand_uniform((R)),hiprand_uniform((R)),hiprand_uniform((R)))

__device__ glm::vec3 randVecUnitSphere(hiprandState *randState) {
    glm::vec3 point;
    do {
        point = 2.0f * RANDVEC3(randState) - glm::vec3(1, 1, 1);
    } while (glm::dot(point, point) >= 1.0f);
    return point;
}

__device__ glm::vec3 reflect(const glm::vec3 &a, const glm::vec3 &b) {
    return a - 2.0f * glm::dot(a, b) * b;
}

Scene::Scene(size_t capacity) : capacity(capacity) {
    catchErr(hipMalloc((void**)&this->volumes, capacity * sizeof(Volume *)));
}

__device__ glm::vec3 Scene::colorAt(const Ray& ray, hiprandState *randState) const {
    Ray r = ray;
    float energy = 1;
    for (size_t b = 0; b < 32; ++b) {
        //Find closest hit
        float closest = FLT_MAX;
        Hit hit;
        for (size_t i = 0; i < capacity; ++i) {
            Hit _hit;
            if (volumes[i]->intersect(r, 0.001f, closest, _hit)) {
                if (_hit.t < closest) {
                    hit = _hit;
                    closest = _hit.t;
                }
            }
        }

        //Bounce if hit
        if (hit.t > 0) {
            // glm::vec3 bounceOut = reflect(glm::normalize(r.b), hit.normal);
            glm::vec3 bounceOut = hit.normal + randVecUnitSphere(randState);
            energy *= 0.5f;
            r.a = hit.point;
            r.b = bounceOut;
        }
        
        //...or we return with sky color
        else {
            const glm::vec3 dir = glm::normalize(r.b);
            const float t = 0.5 * (dir.y + 1.0);
            const glm::vec3 sky = (1.0f - t) * glm::vec3(1, 1, 1) + t * glm::vec3(0.5, 0.7, 1);
            return energy * sky;
        }
    }

    //Energy lost
    return glm::vec3(0, 0, 0);
}
